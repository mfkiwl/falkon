#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>


#define NB 64
#define TILE_DIM 32
#define BLOCK_ROWS 8
#define VEC_MUL_TRIANG_BLOCK_DIM 32


/*
  Matrix is size * size (no support for different size than stride).
  Columns are contiguous.
  The size * size grid is subdivided into NB * size blocks (of rows).
  Each block has NB threads, so each thread copies one row into one
  column (transpose).
  Not a particularly efficient implementation!
*/
template <typename scalar_t>
__global__ void copy_simple_kernel_lower(scalar_t *data, const size_t size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int col_pos = i * size;
        for (int row_pos = i; row_pos < i + i * size; row_pos += size) {
            data[col_pos] = data[row_pos];
            col_pos++;
        }
    }
}

// Same as the _lower version, but we copy dataT to data instead!
template <typename scalar_t>
__global__ void copy_simple_kernel_upper(scalar_t *data, const size_t size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int col_pos = i * size;
        for (int row_pos = i; row_pos < i + i * size; row_pos += size) {
            data[row_pos] = data[col_pos];
            col_pos++;
        }
    }
}


__device__ int2 tri_index_lower(const int linear_index) {
    const int row = (int)((-1 + sqrt((double)(8*linear_index + 1))) / 2.0);
    return make_int2(
        linear_index - row * (row + 1) / 2,
        row
    );
}

__device__ int2 tri_index_upper(const int linear_index) {
    const int row = (int)((-1 + sqrt((double)(8*linear_index + 1))) / 2.0);
    return make_int2(
        row,
        linear_index - row * (row + 1) / 2
    );
}

/* upper = 1, side = 1 (from top) */
template <typename scalar_t>
__global__ void vec_mul_triang_kernel_v1(scalar_t* __restrict__ mat, const scalar_t* __restrict__ vec, const int mat_stride, const int mat_size) {
    const int2 tile_pos = tri_index_upper(blockIdx.x);
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // Check if thread is out of bounds
    const int gx = tile_pos.x * VEC_MUL_TRIANG_BLOCK_DIM + ty;
    const int gy = tile_pos.y * VEC_MUL_TRIANG_BLOCK_DIM + tx;
    if (gy > gx || gx >= mat_size || gy >= mat_size) {
        return;
    }

    // Copy global to register mem
    scalar_t val = mat[gx * mat_stride + gy];
    scalar_t mul = vec[gx];
    // Calc
    val *= mul;
    // Copy back
    mat[gx * mat_stride + gy] = val;
}

/* upper = 1, side = 0 (from left) */
template <typename scalar_t>
__global__ void vec_mul_triang_kernel_v2(scalar_t* __restrict__ mat, const scalar_t* __restrict__ vec, const int mat_stride, const int mat_size) {
    const int2 tile_pos = tri_index_upper(blockIdx.x);
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // Check if thread is out of bounds
    const int gx = tile_pos.x * VEC_MUL_TRIANG_BLOCK_DIM + ty;
    const int gy = tile_pos.y * VEC_MUL_TRIANG_BLOCK_DIM + tx;
    if (gy > gx || gx >= mat_size || gy >= mat_size) {
        return;
    }

    // Copy global to register mem
    scalar_t val = mat[gx * mat_stride + gy];
    scalar_t mul = vec[gy];
    // Calc
    val *= mul;
    // Copy back
    mat[gx * mat_stride + gy] = val;
}

/* upper = 0, side = 1 (from top) */
template <typename scalar_t>
__global__ void vec_mul_triang_kernel_v3(scalar_t* __restrict__ mat, const scalar_t* __restrict__ vec, const int mat_stride, const int mat_size) {
    const int2 tile_pos = tri_index_lower(blockIdx.x);
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // Check if thread is out of bounds
    const int gx = tile_pos.x * VEC_MUL_TRIANG_BLOCK_DIM + ty;
    const int gy = tile_pos.y * VEC_MUL_TRIANG_BLOCK_DIM + tx;
    if (gy < gx || gx >= mat_size || gy >= mat_size) {
        return;
    }

    // Copy global to register mem
    scalar_t val = mat[gx * mat_stride + gy];
    scalar_t mul = vec[gx];
    // Calc
    val *= mul;
    // Copy back
    mat[gx * mat_stride + gy] = val;
}

/* upper = 0, side = 0 (from left) */
template <typename scalar_t>
__global__ void vec_mul_triang_kernel_v4(scalar_t* __restrict__ mat, const scalar_t* __restrict__ vec, const int mat_stride, const int mat_size) {
    const int2 tile_pos = tri_index_lower(blockIdx.x);
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // Check if thread is out of bounds
    const int gx = tile_pos.x * VEC_MUL_TRIANG_BLOCK_DIM + ty;
    const int gy = tile_pos.y * VEC_MUL_TRIANG_BLOCK_DIM + tx;
    if (gy < gx || gx >= mat_size || gy >= mat_size) {
        return;
    }

    // Copy global to register mem
    scalar_t val = mat[gx * mat_stride + gy];
    scalar_t mul = vec[gy];
    // Calc
    val *= mul;
    // Copy back
    mat[gx * mat_stride + gy] = val;
}



template <typename scalar_t>
__global__ void mul_upper_diag(scalar_t *data, const size_t size, const scalar_t mul)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        data += i * size;
        const scalar_t *diag_stop = data + i;
        while (data <= diag_stop) {
            *data *= mul;
            data++;
        }
    }
}


template <typename scalar_t>
__global__ void mul_upper(scalar_t *data, const size_t size, const scalar_t mul)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        data += i * size;
        const scalar_t *diag_stop = data + i;
        while (data < diag_stop) {
            *data *= mul;
            data++;
        }
    }
}


template <typename scalar_t>
__global__ void mul_lower_diag(scalar_t *data, const size_t size, const scalar_t mul)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        data += i * size + i;
        const scalar_t *diag_stop = data + size - i;
        while (data < diag_stop) {
            *data *= mul;
            data++;
        }
    }
}

template <typename scalar_t>
__global__ void mul_lower(scalar_t *data, const size_t size, const scalar_t mul)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        data += i * size + i;
        const scalar_t *diag_stop = data + size - i;
        data++; // Avoid touching the diagonal
        while (data < diag_stop) {
            *data *= mul;
            data++;
        }
    }
}


template<typename scalar_t>
__global__
void matrix_transpose_f(scalar_t * out, const scalar_t * in, const unsigned dim0, const unsigned dim1)
{
    // https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
    // https://arrayfire.com/cuda-optimization-tips-for-matrix-transpose-in-real-world-applications/
    __shared__ scalar_t shrdMem[TILE_DIM][TILE_DIM+1];

    unsigned lx = threadIdx.x;
    unsigned ly = threadIdx.y;

    unsigned gx = lx + TILE_DIM * blockIdx.x;
    unsigned gy = ly + TILE_DIM * blockIdx.y;

#pragma unroll
    for (unsigned repeat = 0; repeat < TILE_DIM; repeat += blockDim.y) {
        unsigned gy_ = gy + repeat;
        if (gx < dim0 && gy_ < dim1) {
            shrdMem[ly + repeat][lx] = in[gy_ * dim0 + gx];
	}
    }
    __syncthreads();

    gx = lx + TILE_DIM * blockIdx.y;
    gy = ly + TILE_DIM * blockIdx.x;

#pragma unroll
    for (unsigned repeat = 0; repeat < TILE_DIM; repeat += blockDim.y) {
        unsigned gy_ = gy + repeat;
        if (gx < dim1 && gy_ < dim0)
            out[gy_ * dim1 + gx] = shrdMem[lx][ly + repeat];
    }
}


template<typename scalar_t>
__global__
void matrix_transpose_c(scalar_t * out, const scalar_t * in, const unsigned dim0, const unsigned dim1)
{
    // https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
    // https://arrayfire.com/cuda-optimization-tips-for-matrix-transpose-in-real-world-applications/
    __shared__ scalar_t shrdMem[TILE_DIM][TILE_DIM+1];

    unsigned lx = threadIdx.x;
    unsigned ly = threadIdx.y;

    unsigned gx = lx + TILE_DIM * blockIdx.x;
    unsigned gy = ly + TILE_DIM * blockIdx.y;

#pragma unroll
    for (unsigned repeat = 0; repeat < TILE_DIM; repeat += blockDim.x) {
	unsigned gx_ = gx + repeat;
        //unsigned gy_ = gy + repeat;
        if (gx_ < dim0 && gy < dim1) {
	    shrdMem[lx + repeat][ly] = in[gx_ * dim1 + gy];
            //shrdMem[ly + repeat][lx] = in[gy_ * dim0 + gx];
	}
    }
    __syncthreads();

    gx = lx + TILE_DIM * blockIdx.y;
    gy = ly + TILE_DIM * blockIdx.x;

#pragma unroll
    for (unsigned repeat = 0; repeat < TILE_DIM; repeat += blockDim.x) {
	unsigned gx_ = gx + repeat;
        //unsigned gy_ = gy + repeat;
        if (gx_ < dim1 && gy < dim0) {
            out[gx_ * dim0 + gy] = shrdMem[ly][lx + repeat];
            //out[gy_ * dim1 + gx] = shrdMem[lx][ly + repeat];
	}
    }
}

int ceildiv(int dividend, int divisor) {
    int res = dividend / divisor;
    if (dividend % divisor != 0)
        res++;
    return res;
}


torch::Tensor cuda_vec_mul_triang(torch::Tensor &A, torch::Tensor &v, bool upper, int side) {
    if (!A.is_cuda())
        AT_ERROR("Input A must be a CUDA tensor.");
    if (!v.is_cuda())
        AT_ERROR("Input v must be a CUDA tensor.");
    if (device_of(v) != device_of(A))
        AT_ERROR("Inputs A, v must be on the same CUDA device.");
    if (A.size(0) != A.size(1))
        AT_ERROR("Input A must be square.");
    if (A.size(0) != v.size(0))
        AT_ERROR("Input v must be of the same dimension as matrix A.");

    int mat_stride = A.stride(1);
    const auto mat_size = A.size(0);
    const auto scalar_type = A.scalar_type();
    // Check matrix contiguity
    bool fortran_contig = false;
    if (A.stride(0) == 1)
        fortran_contig = true;
    // Flip operation if C-contiguous
    bool bside = (bool)side;
    bool bupper = upper;
    if (!fortran_contig) {
        bupper = !upper;
        bside = !bside;
        mat_stride = A.stride(0);
    }

    const int grid_height = ceildiv(mat_size, VEC_MUL_TRIANG_BLOCK_DIM);
    const dim3 dimGrid(grid_height * (grid_height + 1) / 2, 1);
    const dim3 dimBlock(VEC_MUL_TRIANG_BLOCK_DIM, VEC_MUL_TRIANG_BLOCK_DIM);

    AT_DISPATCH_FLOATING_TYPES(scalar_type, "dispatch_vec_mul_triang", [&] {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        at::DeviceGuard g(A.device());
        // Choose correct kernel
        if (bupper && bside)
            vec_mul_triang_kernel_v1<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(
                A.data_ptr<scalar_t>(), v.data_ptr<scalar_t>(), mat_stride, mat_size);
        else if (bupper && !bside)
            vec_mul_triang_kernel_v2<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(
                A.data_ptr<scalar_t>(), v.data_ptr<scalar_t>(), mat_stride, mat_size);
        else if (!bupper && bside)
            vec_mul_triang_kernel_v3<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(
                A.data_ptr<scalar_t>(), v.data_ptr<scalar_t>(), mat_stride, mat_size);
        else if (!bupper && !bside)
            vec_mul_triang_kernel_v4<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(
                A.data_ptr<scalar_t>(), v.data_ptr<scalar_t>(), mat_stride, mat_size);
    });
    return A;
}


torch::Tensor cuda_copy_triang(torch::Tensor &A, bool upper) {
    if (!A.is_cuda()) {
        AT_ERROR("Input A must be a CUDA tensor.");
    }

    bool needs_transpose = false;
    if (A.stride(0) != 1) {
        // Not F-contig (assume C-contig)
        A = torch::transpose(A, 0, 1);
        upper = !upper;
        needs_transpose = true;
    }

    const auto nx = A.size(0);
    const auto ny = A.size(1);
    const auto scalar_type = A.scalar_type();

    const dim3 dimGrid(ceildiv(nx, NB));
    const dim3 dimBlock(NB);

    /* Run CUDA kernel */
    AT_DISPATCH_FLOATING_TYPES(scalar_type, "dispatch", [&] {
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    at::DeviceGuard g(A.device());
    if (upper) {
        copy_simple_kernel_upper<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx);
    } else {
        copy_simple_kernel_lower<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx);
    }
    });

    if (needs_transpose) {
    A = torch::transpose(A, 0, 1);
    }
    return A;
}

torch::Tensor cuda_mul_triang(torch::Tensor &A, bool upper, const bool preserve_diag, const double multiplier) {
    if (!A.is_cuda()) {
        AT_ERROR("Input A must be a CUDA tensor.");
    }
    if (A.stride(0) != 1) {
        upper = !upper;
    }

    const auto nx = A.size(0);
    const auto scalar_type = A.scalar_type();
    const dim3 dimGrid(ceildiv(nx, NB));
    const dim3 dimBlock(NB);

    AT_DISPATCH_FLOATING_TYPES(scalar_type, "dispatch", [&] {
    const scalar_t mul = (scalar_t)multiplier;
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    at::DeviceGuard g(A.device());
    if (upper && preserve_diag) {  // U, preserve
        mul_upper<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx, mul);
    } else if (upper) {            // U, no-preserve
        mul_upper_diag<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx, mul);
    } else if (preserve_diag) {    // L, preserve
        mul_lower<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx, mul);
    } else {                       // L, no-preserve
        mul_lower_diag<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx, mul);
    }
    });
    return A;
}

torch::Tensor cuda_transpose(torch::Tensor &input, torch::Tensor &output) {
    if (!input.is_cuda())
        AT_ERROR("Input must be a CUDA tensor.");
    if (!output.is_cuda())
        AT_ERROR("Output must be a CUDA tensor.");
    if (input.size(0) != output.size(1) || input.size(1) != output.size(0))
        AT_ERROR("Input and output matrices must be of the same size.");
    // TODO: Check strides are consistent

    const auto nx = input.size(0);
    const auto ny = input.size(1);
    const auto scalar_type = input.scalar_type();
    bool fortran_contig = false;
    if (input.stride(0) == 1) {
        fortran_contig = true;
    }

    const dim3 dimGrid(ceildiv(nx, TILE_DIM), ceildiv(ny, TILE_DIM), 1);

    AT_DISPATCH_FLOATING_TYPES(scalar_type, "dispatch", [&] {

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    at::DeviceGuard g(input.device());
    if (fortran_contig) {
        const dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
        matrix_transpose_f<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), nx, ny);
    } else {
        const dim3 dimBlock(BLOCK_ROWS, TILE_DIM, 1);
        matrix_transpose_c<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), nx, ny);
    }

    });
    return output;
}
