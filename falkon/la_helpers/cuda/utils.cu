#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>


#define NB 64
#define TILE_DIM 32
#define BLOCK_ROWS 8


/*
  Matrix is size * size (no support for different size than stride).
  Columns are contiguous.
  The size * size grid is subdivided into NB * size blocks (of rows).
  Each block has NB threads, so each thread copies one row into one
  column (transpose).
  Not a particularly efficient implementation!
*/
template <typename scalar_t>
__global__ void copy_simple_kernel_lower(scalar_t *data, const size_t size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int col_pos = i * size;
        for (int row_pos = i; row_pos < i + i * size; row_pos += size) {
            data[col_pos] = data[row_pos];
            col_pos++;
        }
    }
}

// Same as the _lower version, but we copy dataT to data instead!
template <typename scalar_t>
__global__ void copy_simple_kernel_upper(scalar_t *data, const size_t size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int col_pos = i * size;
        for (int row_pos = i; row_pos < i + i * size; row_pos += size) {
            data[row_pos] = data[col_pos];
            col_pos++;
        }
    }
}


__device__ int2 tri_index_lower(const int linear_index) {
    const int row = (int)((-1 + sqrt((double)(8*linear_index + 1))) / 2.0);
    return make_int2(
        linear_index - row * (row + 1) / 2,
        row
    );
}

__device__ int2 tri_index_upper(const int linear_index) {
    const int row = (int)((-1 + sqrt((double)(8*linear_index + 1))) / 2.0);
    return make_int2(
        row,
        linear_index - row * (row + 1) / 2
    );
}

template <typename scalar_t>
__global__ void vec_mul_triang_kernel_v1(scalar_t* __restrict__ mat, const scalar_t* __restrict__ vec, const int mat_stride) {
    const int2 tile_pos = tri_index_upper(blockIdx.x);
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // Init. shared mem
    __shared__ scalar_t v_seg[blockDim.x];
    __shared__ scalar_t m_tile[blockDim.x][blockDim.y];

    // Copy global to shared mem
    for (int i = 0; i < blockDim.y; i++) {
        m_tile[tx][i] = mat[(tile_pos.x + i) * mat_stride + tile_pos.y + tx]
    }
    v_seg[tx] = vec[tile_pos.x + tx]

    // Calc
    for (int i = 0; i < blockDim.y; i++) {
        m_tile[tx][i] *= v_seg[tx]
    }

    // Copy back (careful about tri-indices)
    for (int i = 0; i < blockDim.y; i++) {
        mat[(tile_pos.x + i) * mat_stride + tile_pos.y + tx] = m_tile[tx][i]
    }
}


template <typename scalar_t>
__global__ void mul_upper_diag(scalar_t *data, const size_t size, const scalar_t mul)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        data += i * size;
        const scalar_t *diag_stop = data + i;
        while (data <= diag_stop) {
            *data *= mul;
            data++;
        }
    }
}


template <typename scalar_t>
__global__ void mul_upper(scalar_t *data, const size_t size, const scalar_t mul)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        data += i * size;
        const scalar_t *diag_stop = data + i;
        while (data < diag_stop) {
            *data *= mul;
            data++;
        }
    }
}


template <typename scalar_t>
__global__ void mul_lower_diag(scalar_t *data, const size_t size, const scalar_t mul)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        data += i * size + i;
        const scalar_t *diag_stop = data + size - i;
        while (data < diag_stop) {
            *data *= mul;
            data++;
        }
    }
}

template <typename scalar_t>
__global__ void mul_lower(scalar_t *data, const size_t size, const scalar_t mul)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        data += i * size + i;
        const scalar_t *diag_stop = data + size - i;
        data++; // Avoid touching the diagonal
        while (data < diag_stop) {
            *data *= mul;
            data++;
        }
    }
}


template<typename scalar_t>
__global__
void matrix_transpose_f(scalar_t * out, const scalar_t * in, const unsigned dim0, const unsigned dim1)
{
    // https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
    // https://arrayfire.com/cuda-optimization-tips-for-matrix-transpose-in-real-world-applications/
    __shared__ scalar_t shrdMem[TILE_DIM][TILE_DIM+1];

    unsigned lx = threadIdx.x;
    unsigned ly = threadIdx.y;

    unsigned gx = lx + TILE_DIM * blockIdx.x;
    unsigned gy = ly + TILE_DIM * blockIdx.y;

#pragma unroll
    for (unsigned repeat = 0; repeat < TILE_DIM; repeat += blockDim.y) {
        unsigned gy_ = gy + repeat;
        if (gx < dim0 && gy_ < dim1) {
            shrdMem[ly + repeat][lx] = in[gy_ * dim0 + gx];
	}
    }
    __syncthreads();

    gx = lx + TILE_DIM * blockIdx.y;
    gy = ly + TILE_DIM * blockIdx.x;

#pragma unroll
    for (unsigned repeat = 0; repeat < TILE_DIM; repeat += blockDim.y) {
        unsigned gy_ = gy + repeat;
        if (gx < dim1 && gy_ < dim0)
            out[gy_ * dim1 + gx] = shrdMem[lx][ly + repeat];
    }
}


template<typename scalar_t>
__global__
void matrix_transpose_c(scalar_t * out, const scalar_t * in, const unsigned dim0, const unsigned dim1)
{
    // https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
    // https://arrayfire.com/cuda-optimization-tips-for-matrix-transpose-in-real-world-applications/
    __shared__ scalar_t shrdMem[TILE_DIM][TILE_DIM+1];

    unsigned lx = threadIdx.x;
    unsigned ly = threadIdx.y;

    unsigned gx = lx + TILE_DIM * blockIdx.x;
    unsigned gy = ly + TILE_DIM * blockIdx.y;

#pragma unroll
    for (unsigned repeat = 0; repeat < TILE_DIM; repeat += blockDim.x) {
	unsigned gx_ = gx + repeat;
        //unsigned gy_ = gy + repeat;
        if (gx_ < dim0 && gy < dim1) {
	    shrdMem[lx + repeat][ly] = in[gx_ * dim1 + gy];
            //shrdMem[ly + repeat][lx] = in[gy_ * dim0 + gx];
	}
    }
    __syncthreads();

    gx = lx + TILE_DIM * blockIdx.y;
    gy = ly + TILE_DIM * blockIdx.x;

#pragma unroll
    for (unsigned repeat = 0; repeat < TILE_DIM; repeat += blockDim.x) {
	unsigned gx_ = gx + repeat;
        //unsigned gy_ = gy + repeat;
        if (gx_ < dim1 && gy < dim0) {
            out[gx_ * dim0 + gy] = shrdMem[ly][lx + repeat];
            //out[gy_ * dim1 + gx] = shrdMem[lx][ly + repeat];
	}
    }
}

int ceildiv(int dividend, int divisor) {
    int res = dividend / divisor;
    if (dividend % divisor != 0)
        res++;
    return res;
}


torch::Tensor cuda_vec_mul_triang(torch::Tensor &A, torch::Tensor &v, bool upper, int side) {
    if (!A.is_cuda()) {
        AT_ERROR("Input A must be a CUDA tensor.");
    }
    if (!v.is_cuda()) {
        AT_ERROR("Input v must be a CUDA tensor.");
    }
    if (device_of(v) != device_of(A)) {
        AT_ERROR("Inputs A, v must be on the same CUDA device.");
    }

    const int block_size = 32;
    const auto nx = A.size(0);
    const auto scalar_type = A.scalar_type();

    const int grid_height = ceildiv(nx, block_size);
    const dim3 dimGrid(grid_height * (grid_height + 1) / 2, 1);
    const dim3 dimBlock(block_size, block_size);

    AT_DISPATCH_FLOATING_TYPES(scalar_type, "dispatch_vec_mul_triang", [&] {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        at::DeviceGuard g(A.device());
        vec_mul_triang_kernel_v1<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(
            A.data_ptr<scalar_t>(), v.data_ptr<scalar_t>(), A.stride(0));
    });
    return A;
}


torch::Tensor cuda_copy_triang(torch::Tensor &A, bool upper) {
    if (!A.is_cuda()) {
        AT_ERROR("Input A must be a CUDA tensor.");
    }

    bool needs_transpose = false;
    if (A.stride(0) != 1) {
        // Not F-contig (assume C-contig)
        A = torch::transpose(A, 0, 1);
        upper = !upper;
        needs_transpose = true;
    }

    const auto nx = A.size(0);
    const auto ny = A.size(1);
    const auto scalar_type = A.scalar_type();

    const dim3 dimGrid(ceildiv(nx, NB));
    const dim3 dimBlock(NB);

    /* Run CUDA kernel */
    AT_DISPATCH_FLOATING_TYPES(scalar_type, "dispatch", [&] {
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    at::DeviceGuard g(A.device());
    if (upper) {
        copy_simple_kernel_upper<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx);
    } else {
        copy_simple_kernel_lower<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx);
    }
    });

    if (needs_transpose) {
    A = torch::transpose(A, 0, 1);
    }
    return A;
}

torch::Tensor cuda_mul_triang(torch::Tensor &A, bool upper, const bool preserve_diag, const double multiplier) {
    if (!A.is_cuda()) {
        AT_ERROR("Input A must be a CUDA tensor.");
    }
    if (A.stride(0) != 1) {
        upper = !upper;
    }

    const auto nx = A.size(0);
    const auto scalar_type = A.scalar_type();
    const dim3 dimGrid(ceildiv(nx, NB));
    const dim3 dimBlock(NB);

    AT_DISPATCH_FLOATING_TYPES(scalar_type, "dispatch", [&] {
    const scalar_t mul = (scalar_t)multiplier;
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    at::DeviceGuard g(A.device());
    if (upper && preserve_diag) {  // U, preserve
        mul_upper<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx, mul);
    } else if (upper) {            // U, no-preserve
        mul_upper_diag<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx, mul);
    } else if (preserve_diag) {    // L, preserve
        mul_lower<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx, mul);
    } else {                       // L, no-preserve
        mul_lower_diag<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(A.data_ptr<scalar_t>(), nx, mul);
    }
    });
    return A;
}

torch::Tensor cuda_transpose(torch::Tensor &input, torch::Tensor &output) {
    if (!input.is_cuda())
        AT_ERROR("Input must be a CUDA tensor.");
    if (!output.is_cuda())
        AT_ERROR("Output must be a CUDA tensor.");
    if (input.size(0) != output.size(1) || input.size(1) != output.size(0))
        AT_ERROR("Input and output matrices must be of the same size.");
    // TODO: Check strides are consistent

    const auto nx = input.size(0);
    const auto ny = input.size(1);
    const auto scalar_type = input.scalar_type();
    bool fortran_contig = false;
    if (input.stride(0) == 1) {
        fortran_contig = true;
    }

    const dim3 dimGrid(ceildiv(nx, TILE_DIM), ceildiv(ny, TILE_DIM), 1);

    AT_DISPATCH_FLOATING_TYPES(scalar_type, "dispatch", [&] {

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    at::DeviceGuard g(input.device());
    if (fortran_contig) {
        const dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
        matrix_transpose_f<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), nx, ny);
    } else {
        const dim3 dimBlock(BLOCK_ROWS, TILE_DIM, 1);
        matrix_transpose_c<scalar_t><<<dimGrid, dimBlock, 0, stream.stream()>>>(
            output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), nx, ny);
    }

    });
    return output;
}
